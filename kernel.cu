#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include "kernel.cuh"

#define getidx int idx=blockIdx.x+threadIdx.x*1024

Real* icudaMalloc(int size) {
	Real* ptr;
	hipMalloc(&ptr, size * sizeof(Real));
	return ptr;
}
void gpuupload(Real* c, Real* g, int size) {
	hipMemcpy(g, c, size * sizeof(Real), hipMemcpyHostToDevice);
}
void gpudownload(Real* c, Real* g, int size) {
	hipMemcpy(c, g, size * sizeof(Real), hipMemcpyDeviceToHost);
}
void gpucpy(Real* g1, Real* g2, int size) {
	hipMemcpy(g2, g1, size * sizeof(Real), hipMemcpyDeviceToDevice);
}

struct Pair {
	int block, thread;
};
Pair blockThreadAlloc(int size) {
	if (size <= 1024) {
		return { 1,size };
	}
	else {
		int blocks = size / 1024 + 1;
		return { blocks,1024 };
	}
}

__global__ void global_add(Real* a, Real* b, Real* res) {
	getidx;
	res[idx] = a[idx] + b[idx];
}
__global__ void global_mul(Real a, Real* vec, Real* res) {
	getidx;
	res[idx] = a * vec[idx];
}

__global__ void global_period(Real* x) {
	getidx;
	if (x[idx] < 0) {
		x[idx] = x[idx] + 1;
	}
	if (x[idx] > 1) {
		x[idx] = x[idx] - 1;
	}
}

__global__ void global_reflect(Real* x, Real* vx) {
	getidx;
	if (x[idx] < 0) {
		x[idx] = -x[idx];
		vx[idx] = -vx[idx];
	}
	if (x[idx] > 1) {
		x[idx] = 2 - x[idx];
		vx[idx] = -vx[idx];
	}
}

__global__ void global_lj(Real* ax, Real* ay, Real* x, Real* y, int size) {
	// constants in LJ potential
	const Real a = 2e-2;
	const Real b = 1e-4;
	const Real max_r2 = 4;
	const Real min_r2 = 1e-4; // prevent too large force
	// calculation
	getidx;
	Real x0 = x[idx];
	Real y0 = y[idx];
	Real fx = 0;
	Real fy = 0;
	for (int i = 0; i < size; i++) {
		if (i == idx)continue;
		Real dx = x[i] - x0;
		Real dy = y[i] - y0;
		Real r2 = dx * dx + dy * dy;
		if (r2 > max_r2)continue;
		if (r2 < min_r2)r2 = min_r2;
		Real a2_over_r2 = a * a / r2;
		Real u = 6 * b / (a * a) * (-2 * pow(a2_over_r2, 7) + pow(a2_over_r2, 4)); // \vec{F}=u\vec{r}
		fx += u * dx;
		fy += u * dy;
	}
	ax[idx] = fx;
	ay[idx] = fy;
}

void cudaAdd(Real* a, Real* b, Real* res, int size) {
	Pair bt = blockThreadAlloc(size);
	global_add << <bt.block, bt.thread >> > (a, b, res);
}
void cudaMul(Real a, Real* vec, Real* res, int size) {
	Pair bt = blockThreadAlloc(size);
	global_mul << <bt.block, bt.thread >> > (a, vec, res);
}

void cudaPeriod(Real* x, int size) {
	Pair bt = blockThreadAlloc(size);
	global_period << <bt.block, bt.thread >> > (x);
}

void cudaReflect(Real* x, Real* vx, int size)
{
	Pair bt = blockThreadAlloc(size);
	global_reflect << <bt.block, bt.thread >> > (x, vx);
}

void cudaLj(Real* ax, Real* ay, Real* x, Real* y, int size) {
	Pair bt = blockThreadAlloc(size);
	global_lj << <bt.block, bt.thread >> > (ax, ay, x, y, size);
}